#include "hip/hip_runtime.h"
#include "MemLeakDebug.h"

#include <cassert>
#include "CudaUtils.h"

#include "ImageFilters.h"

#include <thrust/functional.h>
#include <hipcub/hipcub.hpp>
#include "Vector.h"
#include "SpotDetector.h"

#include "DebugImageCallback.h"
#include "CameraCalibration.h"

#include <memory>

// DLL API
CDLL_EXPORT ISpotDetectorFactory* SpotDetector_Configure(const SpotDetectorConfig& config)
{
	return new SpotDetector::Factory(config);
}
CDLL_EXPORT void SpotDetector_DestroyFactory(ISpotDetectorFactory* factory)
{
	delete factory;
}




struct non_negative
{
	__host__ __device__
		bool operator()(const IndexWithScore &x)
	{
		return x.index >= 0;
	}
};

SpotDetector::SpotDetector(int2 imgsize, const SpotDetectorConfig & cfg) : config(cfg)
{
	temp.Init(imgsize);
	filtered1.Init(imgsize);
	filtered2.Init(imgsize);
	maxFiltered.Init(imgsize);
	int numpixels = imgsize.x*imgsize.y;
	indices.Init(numpixels);
	selectedIndices.Init(numpixels);
	h_selected.Init(numpixels);
	numFoundSpots.Init(1);
	numspots.Init(1);
}




void SpotDetector::Detect(const DeviceImage<float>& srcImage, hipStream_t stream)
{
	UniformFilter2D(srcImage, temp, filtered1, config.uniformFilter1Size, stream);
	UniformFilter2D(srcImage, temp, filtered2, config.uniformFilter2Size, stream);

	ApplyBinaryOperator2D(filtered1, filtered2, filtered1, thrust::minus<float>(), stream);
	ComparisonFilter2D(filtered1, temp, maxFiltered, config.maxFilterSize, thrust::maximum<float>(), stream);

	// convert to indices
	float* d_filtered1 = filtered1.data;
	float* d_max = maxFiltered.data;
	int pitch = filtered1.pitch / sizeof(float);
	IndexWithScore *d_indices = indices.ptr();
	int w = filtered1.width;
	int h = filtered1.height;
	float minIntensity = config.minIntensity, maxIntensity = config.maxIntensity;
	int roisize = config.roisize;

	LaunchKernel(w, h, [=]__device__(int x, int y) {
		float a = d_filtered1[y*pitch + x];
		float b = d_max[y*pitch + x];
		bool isMax = a == b
			&& x > roisize / 2
			&& y > roisize / 2
			&& x + (roisize - roisize / 2) < w - 1
			&& y + (roisize - roisize / 2) < h - 1;

		if (isMax && b > minIntensity && b < maxIntensity)
			d_indices[y*w + x] = { y * w + x, b };
		else
			d_indices[y*w + x] = { -1, 0.0f };
	}, 0, stream);

	if (!partitionTempStorage.ptr())
	{
		size_t tempBytes;
		CUDAErrorCheck(hipcub::DevicePartition::If(0, tempBytes, indices.ptr(), selectedIndices.ptr(), 
			numFoundSpots.ptr(), (int)indices.size(), non_negative(), stream));
		partitionTempStorage.Init(tempBytes);
	}

	size_t tmpsize = partitionTempStorage.size();
	CUDAErrorCheck(hipcub::DevicePartition::If(partitionTempStorage.ptr(), tmpsize, indices.ptr(),
		selectedIndices.ptr(), numFoundSpots.ptr(), (int)indices.size(), non_negative(), stream));

	h_selected.CopyFromDevice(selectedIndices, stream);
	numFoundSpots.CopyToHost(numspots.data(), true, stream);
}

SpotLocationList SpotDetector::GetResults()
{
	SpotLocationList list;
	list.numSpots = numspots[0];
	list.d_indices = selectedIndices.ptr();

	return list;
}



CDLL_EXPORT int SpotDetector_ProcessFrame(const float* frame, int width, int height, int roisize,
	int maxSpots, float* spotScores, Int2* cornerPosYX, float* rois, ISpotDetectorFactory* sdf, IDeviceImageProcessor* calib) 
{
	try {
		DeviceImage<float> img(width, height);
		img.CopyFromHost((const float*)frame);
		PinnedArray<float> h_img(width*height);

		if (calib) {
			calib->ProcessImage(img, 0);
			img.CopyToHost(h_img.data());
			frame = h_img.data();
			hipStreamSynchronize(0);
		}

		ISpotDetector* detector = sdf->CreateInstance(width, height);
		detector->Detect(img, 0);
		hipStreamSynchronize(0);
		detector->Completed();
		auto spotList = detector->GetResults();

		size_t numspots = spotList.numSpots;
		if (numspots > maxSpots) numspots = maxSpots;

		PinnedArray<IndexWithScore> pixelIndices(numspots);
		pixelIndices.CopyFromDevice(spotList.d_indices, numspots, 0);
		delete detector;

		for (int i = 0; i < numspots; i++)
		{
			int centerPixelIndex = pixelIndices[i].index;
			Int2 centerYX = { centerPixelIndex / width, centerPixelIndex % width };
			Int2 corner = centerYX - roisize / 2;
			cornerPosYX[i] = corner;
			spotScores[i] = pixelIndices[i].score;

			for (int y = 0; y < roisize; y++) {
				for (int x = 0; x < roisize; x++) {
					int fy = corner[0] + y, fx = corner[1] + x;
					if (fx < 0) fx = 0;
					if (fy < 0) fy = 0;
					if (fx >= width) fx = width - 1;
					if (fy >= height) fy = height - 1;

					rois[i*roisize*roisize + y * roisize + x] = frame[fy*width + fx];
				}
			}
		}
		return (int)numspots;
	}
	catch (const std::exception& e) {
		DebugPrintf("Exception: %s\n", e.what());
		return 0;
	}

}

CDLL_EXPORT void ExtractROIs(const float *frames, int width, int height, int depth, int roiX, int roiY, int roiZ, 
	const Int3 * startpos, int numspots, float * rois)
{
	for (int i = 0; i < numspots; i++) {
		Int3 pos = startpos[i];
		for (int z = 0; z < roiZ; z++) {
			int fz = pos[0] + z;
			if (fz < 0) fz = 0;
			if (fz >= depth) fz = depth - 1;
			for (int y = 0; y < roiY; y++) {
				int fy = pos[1] + y;
				if (fy >= height) fy = height - 1;
				if (fy < 0) fy = 0;
				for (int x = 0; x < roiX; x++) {
					int fx = pos[2] + x;
					if (fx < 0) fx = 0;
					if (fx >= width) fx = width - 1;

					rois[i*roiX*roiY*roiZ + z * roiX*roiY + y * roiX + x] = frames[z*width*height + fy * width + fx];
				}
			}
		}
	}
}




ISpotDetector * SpotDetector::Factory::CreateInstance(int width, int height)
{
	return new SpotDetector({ width,height }, config);
}


const char * SpotDetector::Factory::GetName()
{
	return "SSA Spot Detector";
}
